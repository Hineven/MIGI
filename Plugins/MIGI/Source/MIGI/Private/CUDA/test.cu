﻿#include "hip/hip_runtime.h"

// Implement a simple cuda kernel to write a string to a buffer.
__global__ void hello(char* a, int* b)
{
	a[threadIdx.x] += b[threadIdx.x];
}
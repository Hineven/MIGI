/*
 * Project MIGINN : MIGINN.cpp
 * Created: 2023/11/13
 * This program is unlicensed. See LICENSE for more.
 */
#include <hip/hip_runtime.h>
#include "MIGINN.h"
#include "MIGINNCUDAHelper.cuh"
#include "MIGINNInternal.cuh"
int MIGIGetCUDAErrorCode() {
    return hipGetLastError();
}

std::string MIGIGetCUDAErrorString() {
    return hipGetErrorString(hipGetLastError());
}

// CUDA context.
hipStream_t GCUDAStream;
// Declare the external semaphore handle and external input & output memory handles.
hipExternalSemaphore_t GExternalSemaphoreHandle;
hipExternalMemory_t GExternalInputMemoryHandle;
hipExternalMemory_t GExternalOutputMemoryHandle;

hipDeviceptr_t GInputBufferAddress;
hipDeviceptr_t GOutputBufferAddress;

std::unique_ptr<MIGINNCacheNetwork> GNetwork;

MIGINNResultType MIGINNInitialize (const MIGINNInitializeParams &Params) {
    // Initialize CUDA context.
    try {
        checkCUDA(cudaInitDevice(Params.InDeviceIndex, 0, 0));
        checkCUDA(hipSetDevice(Params.InDeviceIndex));
        checkCUDA(hipStreamCreate(&GCUDAStream));
    } catch(std::runtime_error & e) {
        return MIGINNResultType::eCUDAError;
    }
    // Import D3D handles into CUDA.
    try {
        // Okay, we can't use C++ 20 due to stupid bugs in fmtlib.
//        hipExternalSemaphoreHandleDesc InExternalSemaphoreHandleDesc{
//                .type = hipExternalSemaphoreHandleTypeD3D12Fence,
//                .handle = {.win32 = {.handle = Params.Platform.Win_D3D12.InD3D12FenceHandle}},
//                .flags = 0
//        };
        hipExternalSemaphoreHandleDesc InExternalSemaphoreHandleDesc{};
        InExternalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeD3D12Fence;
        InExternalSemaphoreHandleDesc.handle.win32.handle = Params.Platform.Win_D3D12.InD3D12FenceHandle;
        InExternalSemaphoreHandleDesc.flags = 0;
        checkCUDA(hipImportExternalSemaphore(&GExternalSemaphoreHandle, &InExternalSemaphoreHandleDesc));
//        hipExternalMemoryHandleDesc InExternalInputMemoryHandleDesc{
//                .type = hipExternalMemoryHandleTypeD3D12Resource,
//                .handle = {.win32 = {.handle = Params.Platform.Win_D3D12.InD3D12InputBufferResourceHandle}},
//                .size = Params.InInputBufferSize,
//                .flags = hipExternalMemoryDedicated
//        };
        hipExternalMemoryHandleDesc InExternalInputMemoryHandleDesc{};
        InExternalInputMemoryHandleDesc.type = hipExternalMemoryHandleTypeD3D12Resource;
        InExternalInputMemoryHandleDesc.handle.win32.handle = Params.Platform.Win_D3D12.InD3D12InputBufferResourceHandle;
        InExternalInputMemoryHandleDesc.size = Params.InInputBufferSize;
        InExternalInputMemoryHandleDesc.flags = hipExternalMemoryDedicated;
        checkCUDA(hipImportExternalMemory(&GExternalInputMemoryHandle, &InExternalInputMemoryHandleDesc));
//        hipExternalMemoryHandleDesc InExternalOutputMemoryHandleDesc{
//                .type = hipExternalMemoryHandleTypeD3D12Resource,
//                .handle = {.win32 = {.handle = Params.Platform.Win_D3D12.InD3D12OutputBufferResourceHandle}},
//                .size = Params.InOutputBufferSize,
//                .flags = hipExternalMemoryDedicated
//        };
        hipExternalMemoryHandleDesc InExternalOutputMemoryHandleDesc{};
        InExternalOutputMemoryHandleDesc.type = hipExternalMemoryHandleTypeD3D12Resource;
        InExternalOutputMemoryHandleDesc.handle.win32.handle = Params.Platform.Win_D3D12.InD3D12OutputBufferResourceHandle;
        InExternalOutputMemoryHandleDesc.size = Params.InOutputBufferSize;
        InExternalOutputMemoryHandleDesc.flags = hipExternalMemoryDedicated;

        checkCUDA(hipImportExternalMemory(&GExternalOutputMemoryHandle, &InExternalOutputMemoryHandleDesc));
        // Get the GPU memory address of the input and output buffers.
//        auto InputBufferDesc = hipExternalMemoryBufferDesc {
//                .offset = Params.InInputBufferOffset,
//                .size = Params.InInputBufferSize,
//                .flags = 0
//        };
        auto InputBufferDesc = hipExternalMemoryBufferDesc{};
        InputBufferDesc.offset = Params.InInputBufferOffset;
        InputBufferDesc.size = Params.InInputBufferSize;
        InputBufferDesc.flags = 0;
        checkCUDA(hipExternalMemoryGetMappedBuffer((void**)&GInputBufferAddress, GExternalInputMemoryHandle, &InputBufferDesc));
//        auto OutputBufferDesc = hipExternalMemoryBufferDesc {
//                .offset = Params.InOutputBufferOffset,
//                .size = Params.InOutputBufferSize,
//                .flags = 0
//        };
        auto OutputBufferDesc = hipExternalMemoryBufferDesc{};
        OutputBufferDesc.offset = Params.InOutputBufferOffset;
        OutputBufferDesc.size = Params.InOutputBufferSize;
        OutputBufferDesc.flags = 0;
        checkCUDA(hipExternalMemoryGetMappedBuffer((void**)&GOutputBufferAddress, GExternalOutputMemoryHandle, &OutputBufferDesc));
    } catch(std::runtime_error & e) {
        return MIGINNResultType::eCUDAError;
    }
    return MIGINNResultType::eSuccess;
}

MIGINNResultType MIGINNDestroy() {
    // Wait for CUDA idle.
    try {
        checkCUDA(hipStreamSynchronize(GCUDAStream));
    } catch(std::runtime_error & e) {
        return MIGINNResultType::eCUDAError;
    }
    // Destroy the CUDA context and release all resources.
    try {
        // Clear pointers
        GInputBufferAddress = 0;
        GOutputBufferAddress = 0;
        // Destroy CUDA context
        checkCUDA(hipStreamDestroy(GCUDAStream));
        GCUDAStream = nullptr;
        checkCUDA(hipDestroyExternalMemory(GExternalInputMemoryHandle));
        GExternalInputMemoryHandle = nullptr;
        checkCUDA(hipDestroyExternalMemory(GExternalOutputMemoryHandle));
        GExternalOutputMemoryHandle = nullptr;
        checkCUDA(hipDestroyExternalSemaphore(GExternalSemaphoreHandle));
        GExternalSemaphoreHandle = nullptr;
        checkCUDA(hipDeviceReset());
    } catch(std::runtime_error & e) {
        return MIGINNResultType::eCUDAError;
    }
    return MIGINNResultType::eSuccess;
}

MIGINNResultType MIGINNWaitFenceValue(uint64_t InWaitFenceValue) {
    // Queue a fence wait in the CUDA stream.
    try {
        auto WaitParams = hipExternalSemaphoreWaitParams{};
        WaitParams.params.fence.value = InWaitFenceValue;
        WaitParams.flags = 0;
        checkCUDA(hipWaitExternalSemaphoresAsync(&GExternalSemaphoreHandle, &WaitParams, 1, GCUDAStream));
    } catch(std::runtime_error & e) {
        return MIGINNResultType::eCUDAError;
    }
    return MIGINNResultType::eSuccess;
}

MIGINNResultType MIGINNSignalFenceValue(uint64_t InSignalFenceValue) {
    // Signal the fence value in the CUDA stream.
    try {
        auto SignalParams = hipExternalSemaphoreSignalParams{};
        SignalParams.params.fence.value = InSignalFenceValue;
        SignalParams.flags = 0;
        checkCUDA(hipSignalExternalSemaphoresAsync(&GExternalSemaphoreHandle, &SignalParams, 1, GCUDAStream));
    } catch(std::runtime_error & e) {
        return MIGINNResultType::eCUDAError;
    }
    return MIGINNResultType::eSuccess;
}

MIGINNResultType MIGINNInitializeNeuralNetwork(const MIGINNNetworkConfig &Config) {
    if(Config.Type == MIGINNNetworkType::eMLP) {
        return (GNetwork = MIGINNMLPCacheNetwork::Create(Config)) ? MIGINNResultType::eSuccess : MIGINNResultType::eError;
    } else return MIGINNResultType::eError;
}


MIGINNResultType MIGINNTrainNetwork(const MIGINNTrainNetworkParams &Params) {
    if(GNetwork) {
        return GNetwork->Train(Params);
    } else return MIGINNResultType::eError;
}

MIGINNResultType MIGINNInference(const MIGINNInferenceParams &Params) {
    if(GNetwork) {
        return GNetwork->Inference(Params);
    } else return MIGINNResultType::eError;
}
